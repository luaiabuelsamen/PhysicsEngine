#include "hip/hip_runtime.h"
// cuda_mass_spring.cu
#include <hip/hip_runtime.h>
#include <iostream>


extern "C" __global__ void simulate_coupled_mass_spring(
    float* positions, float* velocities,
    const float* initialPositions, const float* initialVelocities,
    const float* masses, const float* dampings, const float* springConstants,
    const int* couplingA, const int* couplingB, const float* couplingConstants, int numCouplings,
    int numMasses, int steps, float h)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numMasses) return;

    float x = initialPositions[idx];
    float v = initialVelocities[idx];
    float m = masses[idx];
    float c = dampings[idx];
    float k = springConstants[idx];

    for (int i = 0; i < steps; ++i) {
        // Compute spring force
        float force = -k * x - c * v;
        // Add coupling forces
        for (int j = 0; j < numCouplings; ++j) {
            if (couplingA[j] == idx) {
                force += -couplingConstants[j] * (x - initialPositions[couplingB[j]]);
            } else if (couplingB[j] == idx) {
                force += -couplingConstants[j] * (x - initialPositions[couplingA[j]]);
            }
        }
        float a = force / m;
        v += a * h;
        x += v * h;
    }
    positions[idx] = x;
    velocities[idx] = v;
}

#include "cuda_mass_spring.h"
#include <hip/hip_runtime.h>
#include <vector>

void cuda_coupled_mass_spring(
    float* positions, float* velocities,
    const float* initialPositions, const float* initialVelocities,
    const float* masses, const float* dampings, const float* springConstants,
    const int* couplingA, const int* couplingB, const float* couplingConstants, int numCouplings,
    int numMasses, int steps, float h)
{
    float *d_pos, *d_vel, *d_init_pos, *d_init_vel, *d_masses, *d_dampings, *d_springConstants, *d_couplingConstants;
    int *d_couplingA, *d_couplingB;

    hipMalloc(&d_pos, numMasses * sizeof(float));
    hipMalloc(&d_vel, numMasses * sizeof(float));
    hipMalloc(&d_init_pos, numMasses * sizeof(float));
    hipMalloc(&d_init_vel, numMasses * sizeof(float));
    hipMalloc(&d_masses, numMasses * sizeof(float));
    hipMalloc(&d_dampings, numMasses * sizeof(float));
    hipMalloc(&d_springConstants, numMasses * sizeof(float));
    hipMalloc(&d_couplingConstants, numCouplings * sizeof(float));
    hipMalloc(&d_couplingA, numCouplings * sizeof(int));
    hipMalloc(&d_couplingB, numCouplings * sizeof(int));

    hipMemcpy(d_init_pos, initialPositions, numMasses * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_init_vel, initialVelocities, numMasses * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_masses, masses, numMasses * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dampings, dampings, numMasses * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_springConstants, springConstants, numMasses * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_couplingConstants, couplingConstants, numCouplings * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_couplingA, couplingA, numCouplings * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_couplingB, couplingB, numCouplings * sizeof(int), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (numMasses + threads - 1) / threads;
    simulate_coupled_mass_spring<<<blocks, threads>>>(d_pos, d_vel, d_init_pos, d_init_vel, d_masses, d_dampings, d_springConstants,
        d_couplingA, d_couplingB, d_couplingConstants, numCouplings, numMasses, steps, h);

    hipMemcpy(positions, d_pos, numMasses * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(velocities, d_vel, numMasses * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_pos); hipFree(d_vel); hipFree(d_init_pos); hipFree(d_init_vel);
    hipFree(d_masses); hipFree(d_dampings); hipFree(d_springConstants);
    hipFree(d_couplingConstants); hipFree(d_couplingA); hipFree(d_couplingB);
}


